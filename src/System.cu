#include "hip/hip_runtime.h"
#include "System.h"
#include "SystemStructures.h" 
#include "AreaTriangles.h"
//#include "AreaTrianglesEnergy.h"
#include "BendingTriangles.h"
//#include "BendingTrianglesEnergy.h"
#include "MemRepulsionSprings.h"
#include "MemRepulsionEnergy.h"
#include "LinearSprings.h"
//#include "LinearSpringsEnergy.h"
#include "LJSprings.h"
#include "LJSprings_LJ.h"
#include "NodeAdvance.h"
#include "BucketScheme.h"
#include "Storage.h" 
#include "Edgeswap_test.h"
#include "SystemBuilder.h"
#include <vector>
#include "VolumeComp.h"
#include "VolumeSprings.h"
#include <bits/stdc++.h>
#include "LineTensionSprings.h"
//#include "Growth.h"
#include <math.h>
//#include "SurfaceNormal.h"
//#include "Nodes2Triangles.h"
#include "TurgorForce.h"

///////////////////////////////////////////////////////////////////
///////////////////////// WARNING ////////////////////////////////
//////////////////REMEMBER TO CHANGE THE /////////////////////////
/////////////EQUILIBRIUM LENGTH OF EACH TRIANGLE EDGE /////////////
//////////////// IN THE VECTOR INITIALIZATION ////////////////////
//////////////////SECTION TOWARD THE END OF THE CODE /////////////
////////////////////////////////////////////////////////////////////

 //somehow the gradient is not being set in my version

//bool IsPos (int i){return (i>=0);}
int count_bigger(const std::vector<int>& elems) {
    return std::count_if(elems.begin(), elems.end(), [](int c){return c >= 0;});
}

System::System() {};

void System::Solve_Forces(){

	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);
	
	//setBucketScheme();
	ComputeLinearSprings(
		generalParams, 
		coordInfoVecs,
		linearSpringInfoVecs, 
		ljInfoVecs);
	
	ComputeAreaTriangleSprings(
		
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs);

	ComputeTurgorSprings(
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs
	);

	ComputeCosTriangleSprings(
		
		generalParams,
		coordInfoVecs,  
		bendingTriangleInfoVecs); 
	
	ComputeMemRepulsionSprings(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);

	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs);


	/*ComputeVolumeSprings(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);*/

	/* if (generalParams.true_current_total_volume/initial_volume >= 1.25){
	ComputeLineTensionSprings(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs);
	} */
		
};


void System::solveSystem() {

	std::random_device rand_dev;
	std::mt19937 generator2(rand_dev());
	std::mt19937 generator_edgeswap(rand_dev());

	double MAX_VOLUME_RATIO = 1.5;
	int MAX_GROWTH_NUMBER = 1;
	std::cout<<"MAX_GROWTH_NUMBER (# of edge to expand) = "<<MAX_GROWTH_NUMBER<<std::endl;
	int GROWTH_FREQUENCY = 200;
	std::cout<<"GROWTH_FREQ (how many times Max_Runtime has to be reached to perform growth"<<GROWTH_FREQUENCY<<std::endl;
	double energy_gradient_threshold = 0.01;
	std::cout<<"ENERGY_GRADIENT_THRESHOLD = "<<energy_gradient_threshold<<std::endl;

	generalParams.kT_growth = 1.0;
	generalParams.SCALE_TYPE = 3; 
	// 0:= Gaussian-like weakening
	// 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening
	// 2:= pure Gaussian weakening
	// 3:= isotropic
	// 4:= hill equation
	//Note that (3) is used in combination with sigma = INT_MAX;
	std::cout<<"SCALE TYPE = "<<generalParams.SCALE_TYPE<<std::endl;
	std::cout<<"0:= sigmoidal Gaussian-like weakening, 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening, 2:= pure Gaussian weakening, 3:= isotropic, 4:= hill equation"<<std::endl;
	generalParams.scaling_pow = 2.0;
	std::cout<<"scaling_pow (this is for SCALE_TYPE = 1 case) = "<<generalParams.scaling_pow<<std::endl;
	generalParams.gausssigma = 0.1;
	std::cout<<"gausssigma (this is for the SCALE_TYPE = 0 case) = "<<generalParams.gausssigma<<std::endl;
	//coordInfoVecs.scaling_per_edge.
	//generalParams.hilleqnconst = 0.9;
	//generalParams.hilleqnpow = 40.0;
	std::vector<int> nodes_in_growth;
	std::vector<int> triangles_in_growth;
	std::vector<int> edges_in_growth;
	double dtb; //dtb := distance to boundary
	double dtb_max; //dtb_max := the max distance used to calculate the distance ratio in the Hill equation.
	double sigma = INT_MAX; //if this is set to be INT_MAX then we assume isotropic weakening.
	double sigma_true = sqrt(0.5); //This is the variance used to calculate the scaling of the wall weakening.
	std::cout<<"initial sigma (for gradient distribution variance), based on initial distribution of Cdc42, if using true gaussian weakening = "<<sigma<<std::endl;
	std::cout<<"If sigma = INT_MAX, then we have isotropic weakening scenario"<<std::endl;
	std::cout<<"true sigma (for gaussian-related distribution variance) = "<<sigma_true<<std::endl;

	generalParams.insertion_energy_cost = -log(0.0025);
	std::cout<<"GROWTH: material insertion energy cost (dependent on local chemical concentration) = "<<generalParams.insertion_energy_cost<<std::endl;
	generalParams.strain_threshold = 0.05;//0.01;
	std::cout<<"GROWTH: critical strain threshold used for insertion probability calculation = "<<generalParams.strain_threshold<<std::endl;

	generalParams.growth_energy_scaling = 1.0;//0.01375;
	std::cout<<"GROWTH ENERGY SCALING FOR MATERIAL INSERTION PROBABILITY = "<<generalParams.growth_energy_scaling<<std::endl;
	generalParams.safeguardthreshold = 9;
	std::cout<<"NEIGHBOR SAFE GUARD THRESHOLD = "<<generalParams.safeguardthreshold<<std::endl;
	//safeguardthreshold is the maximum number of neighboring nodes a node can have.

	//////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////
	////////////////////////// PARAMETER SETTINGS ////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////

	double Max_Runtime = generalParams.dt*200.0;
	double Max_RunStep = Max_Runtime/generalParams.dt;
	std::cout<<"Max runtime = "<<Max_Runtime<<std::endl;
	std::cout<<"Max runstep = "<<Max_RunStep<<std::endl;
	bool runSim = true;
	int num_edge_loop;
	double initial_kT;
	initial_kT = generalParams.kT;//This is for the acceptance of change after looping through every edge within proximity.
	double SAMPLE_SIZE = 0.05;
	std::cout<<"Sample ratio: "<<SAMPLE_SIZE<<std::endl;
	std::cout<<"If the Sample raio is 0, it means we have chosen a fixed number of attempt throughout the simulation"<<std::endl;
	//This determines the number of edges to test for bondflip remeshing

	auto edgeswap_ptr = std::make_shared<Edgeswap>(coordInfoVecs, generalParams);
	int RECORD_TIME = 400;//round(Max_RunStep/2);
	std::cout<<"Record frequency = "<<RECORD_TIME<<std::endl;
	//int GROWTH_TIME = 1;
	//std::cout<<"Growth frequency = "<<GROWTH_TIME<<std::endl;
	int translate_frequency = 200;
	std::cout<<"translate + edgeswap frequency = "<<translate_frequency<<std::endl;
	//translate_frequency determines the frequency for the mesh to re-center and perform dynamical remeshing
	int NKBT = GROWTH_FREQUENCY*400;//10000;//7500; //The max number of edge-swap attempt per kBT value
	std::cout<<"Number of edge-swap per kBT value (or total number of edge-swap if kBT is fixed) = "<<NKBT<<std::endl;
	double min_kT = -0.1;//0.21;
	std::cout<<"min kT for simulation termination = "<<min_kT<<std::endl;
	int WHEN = 0;
	double old_total_energy = 0.0;
	double new_total_energy = 0.0;
	double energy_gradient = 0.0;
	double energy_rep = 0.0;
	int Num_of_step_run = 0;
	auto build_ptr = weak_bld_ptr.lock();//upgrade weak builder to access host variables.
	//std::cout<<"initial LJ-x : "<< ljInfoVecs.LJ_PosX <<std::endl;
	//std::cout<<"initial LJ-y : "<< ljInfoVecs.LJ_PosY <<std::endl;
	//std::cout<<"initial LJ-z : "<< ljInfoVecs.LJ_PosZ <<std::endl;
		

    
	double min_energy;
	generalParams.true_num_edges = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
			generalParams.true_num_edges += 1;
		}
	}
	
	//double COMPRESS = 2.0227;
	// double COMPRESS2 = -2.0227;

	/////////////////////////////////////////////////////////////////
	/////////////////////// MEMBRANE RELATED ////////////////////////
	/////////////////////////////////////////////////////////////////
	
	std::vector<double> nodenormal_1(generalParams.maxNodeCount, 0.0);
	std::vector<double> nodenormal_2(generalParams.maxNodeCount, 0.0);
	std::vector<double> nodenormal_3(generalParams.maxNodeCount, 0.0);
	int reduce_counter = 0;

	double VOLUME_FACTOR = 1.0;//2.25;
	//VOLUME_FACTOR determines the target volume which equals to VOLUME_FACTOR*initial_volume.
	//double tip_depth = 0.5;
	//tip_depth is currently unused.

	double LINE_TENSION_THRESHOLD = 0.0;
	std::cout<<"LINE TENSION THRESHOLD for activation of line tension = "<<LINE_TENSION_THRESHOLD<<std::endl;
	double VOLUME_THRESHOLD = 0.0;
	std::cout<<"VOLUME THRESHOLD for activation of weakened membrane = "<<VOLUME_THRESHOLD<<std::endl;
	
	double weakened = 1.90;//6.0;
	//weakened determines the minimum height of the z-coordinate of the membrane node to be considered in the area of weakened mechanical properties.
	//double tip_base = 6.0;
	//tip_base currently unused.

	double EXPAN_THRESHOLD = 0.0;
	double EXPAN_THRESHOLD_weak = 0.0;//1.75;
	std::cout<<"EXPANSION THRESHOLD = "<<EXPAN_THRESHOLD<<std::endl;
	int RULES_OF_EXPAN = 1;	//EXPAN_THRESHOLD is the yielding ratio where a pair of triangles will perform expansion.
	
	std::cout<<"EXPANSION RULE = "<<RULES_OF_EXPAN<<std::endl;
	//EXPAN_THRESHOLD_weak is the secondary yielding ratio.
	//RULES_OF_EXPAN controls how the EXPAN_THRESHOLD is applied:
	// 1:= Both trianglular areas must exceed the threshold value.
	// 2:= If one trianglular area exceeds the treshold value while the other exceeds the secondary threshold value.
	// 3:= If the combined area of the two triangles exceed 2*EXPAN_THRESHOLD.
	// 4:= If a selected edges exceed the threshold value, split the triangles associated with the edge.

	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.centerX += coordInfoVecs.nodeLocX[i];
		generalParams.centerY += coordInfoVecs.nodeLocY[i];
		generalParams.centerZ += coordInfoVecs.nodeLocZ[i];
	}
	generalParams.centerX = generalParams.centerX/generalParams.maxNodeCount;
	generalParams.centerY = generalParams.centerY/generalParams.maxNodeCount;
	generalParams.centerZ = generalParams.centerZ/generalParams.maxNodeCount;
	double displacementX, displacementY, displacementZ;
	double newcenterX, newcenterY, newcenterZ;
	//centerX, centerY, centerZ is determined as the referenced origin for recentering of the mesh.

	std::vector<int> VectorShuffleForGrowthLoop;
	std::vector<int> VectorShuffleForFilamentLoop;
	std::vector<int> VectorShuffleForEdgeswapLoop;

	double max_height = coordInfoVecs.nodeLocZ[35];
	double min_height = coordInfoVecs.nodeLocZ[38];
	int max_height_index = 35;
	/*double max_height = -10000.0;
	int max_height_index = -1;
	std::vector<int> Stiffness_gradient();
    for (int k = 0; k < generalParams.maxNodeCount; k++){
        if (coordInfoVecs. nodeLocZ[k] >= max_height){
			max_height = coordInfoVecs. nodeLocZ[k];
			max_height_index = k;
            }
	}*/
	//Max and min height of the membrane nodes, these have to be changed if the mesh used is changed.

	generalParams.Rmin = 0.301;//0.15;
	//Equilibrium length of an edge of the triangle.
	//generalParams.Rmin_growth = 0.329;
	generalParams.abs_Rmin = 0.301;//0.15;
	//Equilibrium distance between membrane node for volume exclusion.
	areaTriangleInfoVecs.initial_area = 0.039;//0.03927344;//0.009817;
	std::cout<<"equilibrium triangular area = "<<areaTriangleInfoVecs.initial_area<<std::endl;
	//Equilibrium triangular area.
	ljInfoVecs.Rmin_M = 0.0;
	//Equilibrium distance between the nucleus particle and membrane.
	ljInfoVecs.Rcutoff_M = 0.0;
	//Maximal interaction range between the nucleus and membrane.
	ljInfoVecs.Rmin_LJ = 0.0;//3.0//1.0;
	//Equilibrium distance between nuclei.
	ljInfoVecs.Rcutoff_LJ = 0.0;//3.0;//1.0;
	//Maximal interaction range between the nuclei.
	ljInfoVecs.epsilon_M_att1 = 0.0;//6.0;//16.0;
	ljInfoVecs.epsilon_M_att2 = 0.0;//1.0;//1.0;
	std::cout<<"Morse_NM_D_att = "<<ljInfoVecs.epsilon_M_att1<<std::endl;
	std::cout<<"Morse_NM_a_att = "<<ljInfoVecs.epsilon_M_att2<<std::endl;
	//Coefficient for the attractive interaction between nuclei and membrane.
	ljInfoVecs.epsilon_M_rep1 = 0.0;//12.5;//16.0;
	ljInfoVecs.epsilon_M_rep2 = 0.0;//0.5;//1.0;
	std::cout<<"Morse_NM_D_rep = "<<ljInfoVecs.epsilon_M_rep1<<std::endl;
	std::cout<<"Morse_NM_a_rep = "<<ljInfoVecs.epsilon_M_rep2<<std::endl;
	//Coefficient for the repulsive interaction between nuclei and membrane.
	
	ljInfoVecs.epsilon_LJ_rep1 = 0.0;//10.0;//0.5;// 0.06;//7.5;
	ljInfoVecs.epsilon_LJ_rep2 = 0.0;//0.5;//1.0;//1.0;//1.0;
	std::cout<<"Morse_NN_D = "<<ljInfoVecs.epsilon_LJ_rep1<<std::endl;
	std::cout<<"Morse_NN_a = "<<ljInfoVecs.epsilon_LJ_rep2<<std::endl;
	//Coefficient of the interaction between nuclei.

	linearSpringInfoVecs.spring_constant_rep1 = 0.01;
	linearSpringInfoVecs.spring_constant_rep2 = 9.0;
	std::cout<<"Membrane volume exclusion Morse D = "<<linearSpringInfoVecs.spring_constant_rep1<<std::endl;
	std::cout<<"Membrane volume exclusion Morse a = "<<linearSpringInfoVecs.spring_constant_rep2<<std::endl;
	//The coefficient used for non-neighboring membrane node volume exclusion.
	//rep1 is the "D" and rep2 is the "alpha" in the standard form of Morse potential.

	generalParams.volume_spring_constant = 0.2;//(1.0/3.0)*areaTriangleInfoVecs.initial_area*1.0;
	std::cout<<"spring constant for surface normal expansion (pressure within the cell) = "<<generalParams.volume_spring_constant<<std::endl;
	generalParams.line_tension_constant = 50.0;//250.0;
	std::cout<<"spring constant for the septin ring = "<<generalParams.line_tension_constant<<std::endl;
	generalParams.length_scale = 1.0;//0.85;//0.1577;//1.0*generalParams.Rmin;// 0.8333;
	//std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale<<std::endl;

	double scale_linear = linearSpringInfoVecs.spring_constant*0.5;//0.25;//25.0/2.5;//75.0/15.0;
	double scale_bend = bendingTriangleInfoVecs.spring_constant*0.052;//0.05;//10.0/1.0;//75.0/7.5;
	double scale_area = areaTriangleInfoVecs.spring_constant*0.2;//0.25;//50.0/5.0;//75.0/15.0;
	std::cout<<"weakened region linear = "<<scale_linear<<std::endl;
	std::cout<<"weakened region bend = "<<scale_bend<<std::endl;
	std::cout<<"weakened region area = "<<scale_area<<std::endl;
	//linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant/scale_linear;
	//bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant/scale_bend;
	//areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant/scale_area;
	linearSpringInfoVecs.spring_constant_weak = scale_linear;
	bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
	areaTriangleInfoVecs.spring_constant_weak = scale_area;
	//Scaling of the weakend mechanical properties.

	bendingTriangleInfoVecs.initial_angle = 0.087249;//0.04335;
	bendingTriangleInfoVecs.initial_angle_raft = 0.087249;//0.04335;
	bendingTriangleInfoVecs.initial_angle_coat = 0.087249;//0.04335;
	std::cout<<"equilibrium bending angle of the membrane = "<<bendingTriangleInfoVecs.initial_angle<<std::endl;
	//raft and coat are current unused due to the assumption of uniform preferred curvature.
	
	bendingTriangleInfoVecs.spring_constant_raft = 0.0;//bendingTriangleInfoVecs.spring_constant;
	bendingTriangleInfoVecs.spring_constant_coat = 0.0;//bendingTriangleInfoVecs.spring_constant;
	bendingTriangleInfoVecs.spring_constant = bendingTriangleInfoVecs.spring_constant*(2.0/sqrt(3));
	bendingTriangleInfoVecs.spring_constant_raft = bendingTriangleInfoVecs.spring_constant_raft*(2.0/sqrt(3));
	bendingTriangleInfoVecs.spring_constant_coat = bendingTriangleInfoVecs.spring_constant_coat*(2.0/sqrt(3));
	std::cout<<"Effective bending coefficient is calculated by multiplying 2/sqrt(3)"<<std::endl;
	std::cout<<"effective bending coefficient of the membrane = "<<bendingTriangleInfoVecs.spring_constant<<std::endl;
	std::cout<<"effective bending coefficient of the membrane raft = "<<bendingTriangleInfoVecs.spring_constant_raft<<std::endl;
	std::cout<<"effective bending coefficient of the membrane coat = "<<bendingTriangleInfoVecs.spring_constant_coat<<std::endl;

	std::vector<int> pull_nodes_up;// = {35,    76,    79,   111,   113,   151,   153,   360,   361,   362,   363,   364,   365,   505,   506,   515,   516,   593,   632};//{35, 360,   361,   362,   363,   364,   365};
	std::vector<int> pull_nodes_down;// = {38,    86,    89,   121,   123,   144,   146,   378,   379,   380,   381,   382,   383,   535,   536,   545,   546,   602,   626};//{38, 378,   379,   380,   381,   382,   383};
	std::vector<int> push_nodes_down;
	std::vector<int> push_nodes_up;
	for (int i = 0; i < generalParams.maxNodeCount; i++){
		if (coordInfoVecs.nodeLocZ[i] >= 1.43026488631){
			pull_nodes_up.push_back(i);
		}
		if (coordInfoVecs.nodeLocZ[i] <= -1.43026488631){
			pull_nodes_down.push_back(i);
		}
	}

	/////////////////////////////////////////////////////////////////
	////////////////// END OF MEMBRANE RELATED //////////////////////
	/////////////////////////////////////////////////////////////////


	/////////////////////////////////////////////////////////////////
	//////////////////////// NULCEUS RELATED ////////////////////////
	/////////////////////////////////////////////////////////////////
	double beta1 = 0.0;
	double beta2 = 0.0;
	std::cout<<"manual push speed for the nucleus tip = "<<beta1<<std::endl;
	std::cout<<"manual push speed for the remainder of the nucleus = "<<beta2<<std::endl;
	//beta1 is the vertical speed (0, 0, beta1) applied to the nucleus tip.
	//beta2 is the vertical speed (0, 0, beta2) applied to the remainder of the nucleus.

	std::vector<double> V1 = {-0.0};/*, 0.0  ,  0.1966  ,  0.5547 ,  -0.4689 ,   0.2422 ,  -0.2229,
							   -0.4312 ,  -0.0185 ,   0.2887 ,   0.3187 ,   0.7140 ,  
								0.2231 ,  -0.1921 ,	  -0.5541 ,   -0.1542 ,   -0.1689 ,    0.4391 ,
							   -0.6661 ,  -0.6381 ,   0.6256 ,   0.0466 ,  -0.0610 ,   0.5134};
								*/
	std::vector<double> V2 = {0.0};/*, 0.0 ,  -0.4595 ,  -0.4129 ,   0.0954 ,   0.1764 ,   0.4186 ,
							  -0.5602 ,  -0.6082 ,  -0.5318 ,   0.3561 ,   0.0753 ,
							  -0.0917 ,  -0.2596 , 0.2871 ,  -0.3918 ,   0.5195 ,   0.5579 ,
							  -0.2805 ,   0.0133  , -0.0073 ,   0.7426 ,   0.0614 ,  -0.1506};
								*/
	std::vector<double> V3 = { 0.6390};/*, 0.0 ,  -0.5511 ,   0.0267 ,  -0.5240  , -0.4004 ,   0.2850 ,
							   0.2032 ,  -0.1771 ,   0.4048 ,   0.3461 ,  -0.2034 ,
							   0.5041 ,  -0.4535 ,	-0.1241 ,   0.5722 ,  -0.3748 ,  -0.1335 ,
							   -0.0851 ,   0.3213 ,   0.2389 ,   0.0044 ,  -0.7424 ,  -0.7450};
							   */
	//V1, V2, and V3 are the (x,y,z)-coordinate of the nucleus particles.

	for (int i = 0; i < V1.size(); i++){
		ljInfoVecs.LJ_PosX_all.push_back(V1[i]); 
		ljInfoVecs.LJ_PosY_all.push_back(V2[i]);
		ljInfoVecs.LJ_PosZ_all.push_back(V3[i]);
	}  
	
	double NUCLEUS_UPPERHEM_BASE = 0.5;
	double NUCLEUS_LOWERHEM_BASE = -0.6;
	//These values defines the z-coordinate requirement for nucleus particles to be considered tip-region or base-region. This is used to 
	// determine where to apply spring or constant force.

	//////////////////////////////////////////////////////////////////
	///////////////// END OF NUCLEUS RELATED /////////////////////////
	//////////////////////////////////////////////////////////////////

	/*std::vector<int> filament_base(generalParams.maxNodeCountLJ, -1); //= {0,1,2,3,4,5,6,7,8,9,10,11};//{35, 21, 38, etc if we need more points}
	double filament_strength = 0.0;
	double filament_strength_pull = 1.0*filament_strength;
	double filament_Rmin = ((max_height - min_height)/4.0);
	std::cout<<"filament_strength = "<<filament_strength<<std::endl;
	std::cout<<"filament_strength for vertical pull = "<<filament_strength_pull<<std::endl;
	std::cout<<"filament_Rmin = "<<filament_Rmin<<std::endl;
	
	//First, determine the initial membrane nodes having filament bridges
	//with the nuclei particles
	for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
		if (i == 0){
			filament_base[i] = 35;
			continue;
		}
		for (int j = 0; j < generalParams.maxNodeCount; j++){
			double xsquared = (ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j])*
								(ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j]);
			double ysquared = (ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j])*
								(ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j]);
			double zsquared = (ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j])*
								(ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j]);
			double R = sqrt(xsquared + ysquared + zsquared);
			if (R < filament_Rmin*1.1 && j != 35){
				filament_base[i] = j;
				break;
			}
		}
	}*/
	
	//std::vector<double> filament_Rmin;
	//for (int i = 0; i < V3.size();i++){
	//	filament_Rmin.push_back(sqrt((V3[i] - coordInfoVecs.nodeLocZ[38])*(V3[i] - coordInfoVecs.nodeLocZ[38])));
	//}
	//double filament_Rmin = sqrt((V3.back() - coordInfoVecs.nodeLocZ[38])*(V3.back() - coordInfoVecs.nodeLocZ[38]));
	//This part calculates the filament connecting the minimum point (in terms of z-coordinate) to the base of the nuclei cluster.


	//////////////////////////////////////////////////////////////////
	/////////// IDENTIFYING REGIONS WITH DIFFERENT MECH PROP /////////
	//////////////////////////////////////////////////////////////////

	/*ljInfoVecs.forceX_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceY_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceZ_all.reserve(ljInfoVecs.LJ_PosX_all.size());

	generalParams.maxNodeCountLJ = ljInfoVecs.LJ_PosX_all.size();
	std::vector<int> nucleus_in_upperhem(generalParams.maxNodeCountLJ, -1);
	std::vector<int> nucleus_in_lowerhem(generalParams.maxNodeCountLJ, -1);
	for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
		if (ljInfoVecs.LJ_PosZ_all[i] > NUCLEUS_UPPERHEM_BASE){
			nucleus_in_upperhem[i] = 1;
		}
		if (ljInfoVecs.LJ_PosZ_all[i] < NUCLEUS_LOWERHEM_BASE){
			nucleus_in_lowerhem[i] = 1;
		}
	}*/
	

	std::vector<int> out;
	//int ALPHA;

	std::vector<bool> boundary_edges;
	boundary_edges.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Triangles_1[i] == coordInfoVecs.edges2Triangles_2[i]){
			boundary_edges.push_back(true);
		}
		else {
			boundary_edges.push_back(false);
		}
	}

	std::vector<int> edgeIndices;
	edgeIndices.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; ++i){
		//edgeIndices.push_back(edge_to_ljparticle[i]);
		if (boundary_edges[i] == false){
			edgeIndices.push_back(i);
		}
		else {
			edgeIndices.push_back(-1);
		}
	}



	auto it = remove_if(edgeIndices.begin(), edgeIndices.end(),  [](const int i) {return i < 0; });
	edgeIndices.erase(it, edgeIndices.end());
	
	std::vector<int> row2 = {35 ,   76 ,   79 ,  111 ,  113 ,  151 ,  153 ,  360 ,  361 ,  362 ,  363 ,  364 ,  365 ,  505 ,  506 ,  515 ,  516 ,  593 ,  632};
	//std::vector<int> nodes_to_center;
	//generalParams.nodes_in_upperhem.resize(generalParams.maxNodeCount,-1);

	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.nodes_in_upperhem[i] = -1;
	}

	for (int i = 0; i < row2.size(); i++){
		generalParams.nodes_in_upperhem[row2[i]] = 1;
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	}
	// for (int i = 0; i < generalParams.maxNodeCount; i++){
	// 	if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + weakened)){
	// 		generalParams.nodes_in_upperhem[i] = 1;
	// 	}
	// 	else{
	// 		generalParams.nodes_in_upperhem[i] = -1;
	// 	}
	// //	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	// }

	//std::vector<int> nodes_to_center;
	//std::vector<int> nodes_in_tip;
	//nodes_in_tip.resize(generalParams.maxNodeCount);
	//for (int i = 0; i < generalParams.maxNodeCount; i++){
	//	if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + tip_base)){
	//		nodes_in_tip[i] = 1;
	//	}
	//	else{
	//		nodes_in_tip[i] = -1;
	//	}
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	//}

	//generalParams.triangles_in_upperhem.resize(coordInfoVecs.num_triangles);
	for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		int aaa = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_1[i]];
		//std::cout<<aaa<<std::endl;
		int bbb = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_2[i]];
		//std::cout<<bbb<<std::endl;
		int ccc = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_3[i]];
		//std::cout<<ccc<<std::endl;
		if ((aaa+bbb+ccc)==3){
			generalParams.triangles_in_upperhem[i] = 1;
			//triangles_in_upperhem.push_back(i);
		}
		//else if ((aaa+bbb+ccc)==1){
		//	generalParams.triangles_in_upperhem[i] = 0;
			//triangles_in_upperhem.push_back(i);
		//}
		else{
			generalParams.triangles_in_upperhem[i] = -1;
		}
	//	std::cout<<"triangle "<<i<<" "<<generalParams.triangles_in_upperhem[i]<<std::endl;		
	}

	//std::vector<int> edges_in_upperhem;
//	generalParams.edges_in_upperhem.resize(coordInfoVecs.num_edges);
	int edges_in_upperhem_COUNT = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		int aaa = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_1[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_1[i]];
		int bbb = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_2[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_2[i]];
		if (aaa == 1 && bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			//generalParams.edges_in_upperhem_list.push_back(i);
			generalParams.edges_in_upperhem_list[i] = i;
			edges_in_upperhem_COUNT += 1;
		}
		else if (aaa == 1 || bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
			edges_in_upperhem_COUNT += 1;
		}
		else{
			generalParams.edges_in_upperhem[i] = -1;
			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
		}
		
	}
	std::cout<<"INITIAL EDGES IN UPPERHEM = "<<edges_in_upperhem_COUNT<<std::endl;

	int COUNTING_EDGE = 0;
	for (int y = 0; y < coordInfoVecs.num_edges; y++){
		if (generalParams.edges_in_upperhem_list[y] >= 0){
			COUNTING_EDGE += 1;
		}
		generalParams.edges_in_upperhem_list_length = COUNTING_EDGE;
	}
	

	//Find the boundary of the nodes_in_upperhem region
	//generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
	std::vector<int> boundary_node_list;
	std::vector<int> boundary_edge_list;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		double T1 = coordInfoVecs.edges2Triangles_1[i];
		double T2 = coordInfoVecs.edges2Triangles_2[i];
		if (generalParams.triangles_in_upperhem[T1] == 1 && generalParams.triangles_in_upperhem[T2] != 1){
			generalParams.boundaries_in_upperhem[i] = 1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		//	generalParams.triangles_in_upperhem[T1] = 0;
		//	generalParams.triangles_in_upperhem[T2] = 0;
			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
			boundary_node_list.push_back(bdry_node1);
			boundary_node_list.push_back(bdry_node2);
			boundary_edge_list.push_back(i);
			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
			//coordInfoVecs.isNodeFixed[bdry_node1] = true;
			//coordInfoVecs.isNodeFixed[bdry_node2] = true;
		}
		else if (generalParams.triangles_in_upperhem[T1] != 1 && generalParams.triangles_in_upperhem[T2] == 1){
			generalParams.boundaries_in_upperhem[i] = 1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		//	generalParams.triangles_in_upperhem[T1] = 0;
		//	generalParams.triangles_in_upperhem[T2] = 0;
			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
			boundary_node_list.push_back(bdry_node1);
			boundary_node_list.push_back(bdry_node2);
			boundary_edge_list.push_back(i);
			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
			//coordInfoVecs.isNodeFixed[bdry_node1] = true;
			//coordInfoVecs.isNodeFixed[bdry_node2] = true;
		}
		else {
			generalParams.boundaries_in_upperhem[i] = -1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		}
	}
	std::cout<<"size of boundary_node_list (this is double-counted) = "<<boundary_node_list.size()<<std::endl;
	//generalParams.eq_total_boundary_length = generalParams.boundaries_in_upperhem.size()*generalParams.Rmin;

	/*for (int i = 0; i < coordInfoVecs.num_edges; i++){
		int aaa = coordInfoVecs.edges2Nodes_1[i];
		int bbb = coordInfoVecs.edges2Nodes_2[i];
		if (aaa == 1 && bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			generalParams.edges_in_upperhem_list.push_back(i);
		}
		else if (aaa == 1 || bbb == 1){
			generalParams.edges_in_upperhem[i] = 0;
		}
		else{
			generalParams.edges_in_upperhem[i] = -1;
		}
		
	}*/
	
	

	int true_num_edges_in_upperhem = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
		true_num_edges_in_upperhem += 1;
		}
	}
	

	//std::vector<int> edge_to_ljparticle;
	//generalParams.edge_to_ljparticle.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		generalParams.edge_to_ljparticle.push_back(-1);
	};
	/////////////////////////////////////////////////////////////////////
	////////////// END OF IDENTIFYING REG. WITH DIFF. MECH PROP /////////
	/////////////////////////////////////////////////////////////////////


	//std::cout<<"ERROR HERE?"<<std::endl;
	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs
	);
	//std::cout<<"ERROR HERE 2?"<<std::endl;
	double initial_volume;
	initial_volume = generalParams.true_current_total_volume;
	generalParams.eq_total_volume = generalParams.true_current_total_volume*VOLUME_FACTOR;//This is for setting different equilibrium volume to mimic growth or shirnkage.
	std::cout<<"true_current_total_volume = "<<generalParams.true_current_total_volume<<std::endl;
	std::cout<<"eq_total_volume = "<<generalParams.eq_total_volume<<std::endl;

	//////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////// START OF ACTUAL SIMULATION /////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////

	/* Build the initial gradient weakend scale */
	dtb = 0.0;//dtb := distance to boundary
	generalParams.septin_ring_z = 0.0;
	generalParams.boundary_z = 0.0;
	//for (int k = 0; k < boundary_edge_list.size(); k++){
	for (int k = 0; k < boundary_node_list.size(); k++){
		double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
		//double n2 = coordInfoVecs.edges2Nodes_2[boundary_edge_list[k]];
		//double cent_of_edge_x = (coordInfoVecs.nodeLocX[n1] + coordInfoVecs.nodeLocX[n2])/2.0;
		//double cent_of_edge_y = (coordInfoVecs.nodeLocY[n1] + coordInfoVecs.nodeLocY[n2])/2.0;
		//double cent_of_edge_z = (coordInfoVecs.nodeLocZ[n1] + coordInfoVecs.nodeLocZ[n2])/2.0;
		double dist_x = coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1];//cent_of_edge_x;
		double dist_y = coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1];//cent_of_edge_y;
		double dist_z = coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1];//cent_of_edge_z;
		double temp_dist = sqrt((coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1])*(coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1]) +
		(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1])*(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1]) +
			(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1])*(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1]));
		generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
		if (temp_dist >= dtb){
			dtb = temp_dist;
			/* "dtb" will be used to identify where the septin ring is located, and used to determine the Hill coefficient*/
		}
	}
	std::cout<<"dtb = "<<dtb<<std::endl;
	//generalParams.septin_ring_z = generalParams.septin_ring_z/boundary_node_list.size();
	//generalParams.boundary_z = generalParams.septin_ring_z - generalParams.Rmin;
	/* dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift */
	dtb_max = dtb + (generalParams.Rmin);
	
	std::cout<<"initial distance between cell tip and the boundary of weakened area = "<<dtb<<std::endl;
	std::cout<<"Notice that here, the distance from the tip to the boundary is slightly extended by half of the equilibrium length of an edge"<<std::endl;
	//std::cout<<"If this message is present, we are forcing a fixed portion of the bud tip to be occupied by the max concentration"<<std::endl;
	generalParams.hilleqnconst = (dtb + generalParams.Rmin/4.0)/dtb_max;
	//generalParams.hilleqnconst = dtb/dtb_max;
	generalParams.hilleqnpow = 17.0;
	std::cout<<"hill equation constant K = "<<generalParams.hilleqnconst<<std::endl;
	std::cout<<"hill (equation) coefficient = "<<generalParams.hilleqnpow<<std::endl;
	std::cout<<"NOTE: IN THIS SIMULATION, THE LOCATION WHERE 50% WEAKENING IS EXPERIENCED IS LOCATED SLIGHTLY AWAY FROM THE SEPTIN RING, "<<std::endl;
	std::cout<<"THIS IS DUE TO THE FACT THAT IN ISOTROPIC CASE, SEPTIN RING LOCATION MUST BE SUFFICIENTLY WEAKENED TO INDUCE BUDDING"<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;
	std::cout<<" "<<std::endl;


	edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
	edgeswap_ptr->gradient_weakening_update_host_vecs(sigma,
		max_height_index,
		dtb,
		dtb_max,
		generalParams,
		coordInfoVecs,
		build_ptr->hostSetInfoVecs);
	for (int u = 0; u < generalParams.maxNodeCount; u++){
		int BETA = edgeswap_ptr->nodes2Triangles_host_vecs(
			u,
			build_ptr->hostSetInfoVecs,
			coordInfoVecs,
			generalParams,
			auxVecs);
	}
	edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
	/*for (int h = 0; h < coordInfoVecs.num_edges; h++){
		std::cout<<coordInfoVecs.scaling_per_edge[h]<<std::endl;
		double scaling = 0.0;//(spring_constant_weak/spring_constant);
		double what_spring_constant = bendingTriangleInfoVecs.spring_constant*((1.0/(1.0+pow(0.5/coordInfoVecs.scaling_per_edge[h], 6.0)))*(1-scaling) + scaling);
		if (what_spring_constant < bendingTriangleInfoVecs.spring_constant_weak){what_spring_constant = bendingTriangleInfoVecs.spring_constant_weak;}
		std::cout<<"bend_constant = "<<what_spring_constant<<std::endl;
	}
	std::cout<<"end of scaling_per_edge printout"<<std::endl;*/
	
	
	
	/*for (int i = 0; i < 10; i++){
		std::cout<<"nodess2Triangles_1[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_1[i]<<std::endl;
		std::cout<<"nodess2Triangles_2[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_2[i]<<std::endl;
		std::cout<<"nodess2Triangles_3[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_3[i]<<std::endl;
		std::cout<<"nodess2Triangles_4[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_4[i]<<std::endl;
		std::cout<<"nodess2Triangles_5[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_5[i]<<std::endl;
		std::cout<<"nodess2Triangles_6[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_6[i]<<std::endl;
		std::cout<<"nodess2Triangles_7[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_7[i]<<std::endl;
		std::cout<<"nodess2Triangles_8[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_8[i]<<std::endl;
		std::cout<<"nodess2Triangles_9[ "<<i<<" ] = "<<coordInfoVecs.nodes2Triangles_9[i]<<std::endl;

	}*/
	while (runSim == true){
		//WHEN += 1;
		double current_time = 0.0;
		//nodenormal_1.resize(generalParams.maxNodeCount);
		//nodenormal_2.resize(generalParams.maxNodeCount);
		//nodenormal_3.resize(generalParams.maxNodeCount);
		//std::fill(nodenormal_1.begin(), nodenormal_1.end(), 0.0);
		//std::fill(nodenormal_2.begin(), nodenormal_2.end(), 0.0);
		//std::fill(nodenormal_3.begin(), nodenormal_3.end(), 0.0);
		/* for (int k = 0; k < coordInfoVecs.num_triangles; k++){
			if (coordInfoVecs.triangles2Nodes_1[k] != INT_MAX || coordInfoVecs.triangles2Nodes_3[k] != INT_MAX || coordInfoVecs.triangles2Nodes_3[k] != INT_MAX){
				double x1 = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
				double y1 = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
				double z1 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
				double x2 = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
				double y2 = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
				double z2 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
				double x3 = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
				double y3 = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
				double z3 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
				double nx = (y2 - y1)*(z3 - z1) - (y3 - y1)*(z2 - z1);
				double ny = -((x2 - x1)*(z3 - z1) - (x3 - x1)*(z2 - z1));
				double nz = (x2 - x1)*(y3 - y1) - (x3 - x1)*(y2 - y1);
				nodenormal_1[coordInfoVecs.triangles2Nodes_1[k]] += nx;
				nodenormal_2[coordInfoVecs.triangles2Nodes_1[k]] += ny;
				nodenormal_3[coordInfoVecs.triangles2Nodes_1[k]] += nz;
				nodenormal_1[coordInfoVecs.triangles2Nodes_2[k]] += nx;
				nodenormal_2[coordInfoVecs.triangles2Nodes_2[k]] += ny;
				nodenormal_3[coordInfoVecs.triangles2Nodes_2[k]] += nz;
				nodenormal_1[coordInfoVecs.triangles2Nodes_3[k]] += nx;
				nodenormal_2[coordInfoVecs.triangles2Nodes_3[k]] += ny;
				nodenormal_3[coordInfoVecs.triangles2Nodes_3[k]] += nz;
			}
			else{continue;}
		}
		for (int k = 0; k < nodenormal_1.size(); k++){
			double UN = sqrt((nodenormal_1[k]*nodenormal_1[k]) + (nodenormal_2[k]*nodenormal_2[k]) + (nodenormal_3[k]*nodenormal_3[k]));
			nodenormal_1[k] = nodenormal_1[k]/UN;
			nodenormal_2[k] = nodenormal_2[k]/UN;
			nodenormal_3[k] = nodenormal_3[k]/UN;
		} */


		
		//generalParams.kT = 1.0;//reset kT before simulations starts.
		//Max_Runtime = 0.0;//2.5;
		int translate_counter = 0;
		
			while (current_time < 0.0*(Max_Runtime)){
					translate_counter += 1;
					Solve_Forces();
					//if (generalParams.true_current_total_volume/initial_volume >= LINE_TENSION_THRESHOLD){
					//	ComputeLineTensionSprings(
					//		generalParams,
					//		coordInfoVecs,
					//		linearSpringInfoVecs);
					//	}
				
					energy_rep =
					ComputeMemRepulsionEnergy(
						coordInfoVecs,
						linearSpringInfoVecs, 
						capsidInfoVecs,
						generalParams,
						auxVecs);

					//now forces are computed, move nodes.
					
					

					

					double beta;
					

				/*for (int k = 0; k < generalParams.maxNodeCount; k++){
					if (generalParams.nodes_in_upperhem[k] == 1 && generalParams.boundaries_in_upperhem[k] != 1){
						coordInfoVecs.nodeForceX[k] = 0.001*coordInfoVecs.nodeForceX[k];
						coordInfoVecs.nodeForceY[k] = 0.001*coordInfoVecs.nodeForceY[k];
						coordInfoVecs.nodeForceZ[k] = 0.001*coordInfoVecs.nodeForceZ[k];
					}
				}*/

				/*for (int k = 0; k < generalParams.maxNodeCount; k++){
					coordInfoVecs.nodeForceX[k] += generalParams.volume_spring_constant*coordInfoVecs.SurfaceNormalX[k];
					coordInfoVecs.nodeForceY[k] += generalParams.volume_spring_constant*coordInfoVecs.SurfaceNormalY[k];
					coordInfoVecs.nodeForceZ[k] += generalParams.volume_spring_constant*coordInfoVecs.SurfaceNormalZ[k];
				}*/

				/*ComputeSurfaceNormal(
					coordInfoVecs,
					generalParams,
					auxVecs
				);*/
				/*for (int u = 0; u < generalParams.maxNodeCount; u++){
					int GAMMA = edgeswap_ptr->surfaceNormal_device_vecs(
						u,
						coordInfoVecs,
						generalParams
					);
				}*/
				AdvancePositions(
					coordInfoVecs,
					generalParams,
					domainParams);

/*				if (translate_counter % translate_frequency == 1){

					newcenterX = 0.0;
					newcenterY = 0.0;
					newcenterZ = 0.0;
					for (int i = 0; i < generalParams.maxNodeCount; i++){//for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
						newcenterX += coordInfoVecs.nodeLocX[i];
						newcenterY += coordInfoVecs.nodeLocY[i];
						newcenterZ += coordInfoVecs.nodeLocZ[i];
					}
					newcenterX = newcenterX/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					newcenterY = newcenterY/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					newcenterZ = newcenterZ/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					displacementX = newcenterX - generalParams.centerX;
					displacementY = newcenterY - generalParams.centerY;
					displacementZ = newcenterZ - generalParams.centerZ;

					for (int i = 0; i < generalParams.maxNodeCount; i++){
						coordInfoVecs.nodeLocX[i] += -displacementX;
						coordInfoVecs.nodeLocY[i] += -displacementY;
						coordInfoVecs.nodeLocZ[i] += -displacementZ;
					}
					for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
						ljInfoVecs.LJ_PosX_all[i] += -displacementX;
						ljInfoVecs.LJ_PosY_all[i] += -displacementY;
						ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
					}

					//Here we re-establish the new filament base according to the current location of nuclei nodes
					 int maxElementIndex = std::max_element(coordInfoVecs.nodeLocZ.begin(),coordInfoVecs.nodeLocZ.end()) - coordInfoVecs.nodeLocZ.begin();
					for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
						if (i == 0){
							filament_base[i] = maxElementIndex;
							continue;
						}
						for (int j = 0; j < generalParams.maxNodeCount; j++){
							double xsquared = (ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j])*
												(ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j]);
							double ysquared = (ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j])*
												(ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j]);
							double zsquared = (ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j])*
												(ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j]);
							double R = sqrt(xsquared + ysquared + zsquared);
							if (R < (max_height - min_height)/2.0 && j != maxElementIndex){
								filament_base[i] = j;
								break;
							}
							else{filament_base[i] = -1;}
						}
					} 
				}*/
							
					new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
						areaTriangleInfoVecs.area_triangle_energy + 
						bendingTriangleInfoVecs.bending_triangle_energy + 
						0.5*energy_rep;// + 
						//ljInfoVecs.lj_energy_M +
						//ljInfoVecs.lj_energy_LJ +
						//generalParams.volume_energy;

				energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy));
				if (current_time >= Max_Runtime*0.25 && energy_gradient < energy_gradient_threshold){
					break;
				}
				old_total_energy = new_total_energy;
				current_time+=generalParams.dt;
				

			}
			
		   
			/*max_height = -10000.0;
			min_height = 10000.0;
			for (int k = 0; k < generalParams.maxNodeCount; k++){
				if (coordInfoVecs. nodeLocZ[k] >= max_height){
					max_height = coordInfoVecs. nodeLocZ[k];
				}
				if (coordInfoVecs.nodeLocZ[k] <= min_height){
					min_height = coordInfoVecs.nodeLocZ[k];
				}
			}*/

		std::cout<<"current time (1st iter before edgeswap): "<< current_time << std::endl;
		std::cout<<"current total energy (1st iter before edgeswap) = "<<new_total_energy<<std::endl;
		std::cout<<"LINEAR ENERGY = "<<linearSpringInfoVecs.linear_spring_energy<<std::endl;
		std::cout<<"BEND ENERGY = "<<bendingTriangleInfoVecs.bending_triangle_energy<<std::endl;
		std::cout<<"AREA ENERGY = "<<areaTriangleInfoVecs.area_triangle_energy<<std::endl;
		std::cout<<"REPULSION ENERGY = "<<energy_rep<<std::endl;
		std::cout<<"VOLUME ENERGY = "<<generalParams.volume_energy<<std::endl;
		std::cout<<"true_current_total_volume = "<<generalParams.true_current_total_volume<<std::endl;
		std::cout<<"eq_total_volume = "<<generalParams.eq_total_volume<<std::endl;
		std::cout<<"current KBT = "<<generalParams.kT<<std::endl;
		if (isnan(new_total_energy)==1){
			std::cout<<"Nan or Inf position update !!!!"<<std::endl;
			runSim = false;
			break;
		}
	
		//edgeswap_ptr->transferDtoH(coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
		storage->print_VTK_File();
		//storage->storeVariables();
		//runSim = false;
		//break;

		int edgeswap_iteration = 0;
		//double preswap_energy = new_total_energy;
		//double postswap_energy;
		//double Ediff = 0.0;
		//initial_kT = generalParams.kT;
		num_edge_loop = 0;//round(true_num_edges_in_upperhem*SAMPLE_SIZE);
		//if (num_edge_loop == 0){
		//	num_edge_loop = 1;
		//}	
		
		std::cout<<"if SAMPLE_SIZE = 0, this implies that all edges in the budding site will be tested for edgeswap"<<std::endl;
		

		

		int LINE_TENSION_START = 0;
		
		bool WEAKENED_START = false;
		bool EDGESWAP_ALGORITHM_TRIGGERED;
 		while (initial_kT > 0){
			
			/*if (generalParams.true_current_total_volume/initial_volume >= VOLUME_THRESHOLD && WEAKENED_START == false){
				linearSpringInfoVecs.spring_constant_weak = scale_linear;
				bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
				areaTriangleInfoVecs.spring_constant_weak = scale_area;
				std::cout<<"membrane weakening initiated"<<std::endl;
				WEAKENED_START = true;
			}*/
 					////////////////////NOW RELAX THE ATTEMPTED EDGESWAP//////////////////////
					 current_time = 0.0;
					 translate_counter = 0;
					 double VOLUME_RATIO = generalParams.true_current_total_volume/generalParams.eq_total_volume;
					//if (VOLUME_RATIO > 0.75 && VOLUME_FACTOR <= 2.5){
					//	VOLUME_FACTOR += 0.2;
					//	generalParams.eq_total_volume = initial_volume*VOLUME_FACTOR;
					//};
					if (generalParams.true_current_total_volume/initial_volume >= LINE_TENSION_THRESHOLD){
						if (LINE_TENSION_START < 1){
							double DIST = 0.0;
							double COUNT = 0.0;
							for (int t = 0; t < coordInfoVecs.num_edges; t++){
								if (generalParams.boundaries_in_upperhem[t] == 1){
									COUNT += 1.0;
									int node1 = coordInfoVecs.edges2Nodes_1[t];
									int node2 = coordInfoVecs.edges2Nodes_2[t];
									DIST += sqrt((coordInfoVecs.nodeLocX[node2] - coordInfoVecs.nodeLocX[node1])*(coordInfoVecs.nodeLocX[node2] - coordInfoVecs.nodeLocX[node1]) +
									(coordInfoVecs.nodeLocY[node2] - coordInfoVecs.nodeLocY[node1])*(coordInfoVecs.nodeLocY[node2] - coordInfoVecs.nodeLocY[node1]) + 
									(coordInfoVecs.nodeLocZ[node2] - coordInfoVecs.nodeLocZ[node1])*(coordInfoVecs.nodeLocZ[node2] - coordInfoVecs.nodeLocZ[node1]));
								}
							}
							generalParams.length_scale = (DIST/COUNT)/generalParams.Rmin;
							std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale*generalParams.Rmin<<std::endl;
							generalParams.eq_total_boundary_length = COUNT*generalParams.length_scale* generalParams.Rmin;
							std::cout<<"equilibrium length of the septin ring = "<<generalParams.eq_total_boundary_length<<std::endl;
							LINE_TENSION_START += 1;
						}
						
					}
					//std::cout<<"start relaxation step"<<std::endl;
					EDGESWAP_ALGORITHM_TRIGGERED = false;
					bool end_of_relaxation = false;
 					while (current_time < Max_Runtime){
						
						 if (Max_Runtime <= 0.0){
							 std::cout<<"Max_Runtime is set to be 0 or negative! "<<std::endl;
							 break;
						 }
						 
						 translate_counter += 1;
						 //std::cout<<"STOPPED BEFORE Solve_Forces"<<std::endl;
						 Solve_Forces();
						 //if (generalParams.true_current_total_volume/initial_volume >= LINE_TENSION_THRESHOLD){
						if (LINE_TENSION_START >= 1){
							ComputeLineTensionSprings(
								generalParams,
								coordInfoVecs,
								linearSpringInfoVecs);
							}
						//std::cout<<"STOPPED BEFORE MemRepul"<<std::endl;
 						energy_rep =
 						ComputeMemRepulsionEnergy(
 							coordInfoVecs,
 							linearSpringInfoVecs, 
 							capsidInfoVecs,
 							generalParams,
							 auxVecs);
					if ((generalParams.true_current_total_volume/initial_volume) < 0.6 || generalParams.true_current_total_volume/initial_volume >= MAX_VOLUME_RATIO){
						generalParams.true_num_edges = 0;
						for (int i = 0; i < coordInfoVecs.num_edges; i++){
							if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
								generalParams.true_num_edges += 1;
							}
						}
						storage-> print_VTK_File();
						storage-> storeVariables();
						runSim = false;
						initial_kT = -0.00000000000000001;
						if (generalParams.true_current_total_volume/initial_volume < 0.6){
							std::cout<<"Cell over compression 60%"<<std::endl;
						}
						else if (generalParams.true_current_total_volume/initial_volume >= MAX_VOLUME_RATIO){
							std::cout<<"Target volume ratio exceeded. Current volume ratio = "<<generalParams.true_current_total_volume/initial_volume<<std::endl;
						}

						Max_Runtime = 0.0;
						runSim = false;
						initial_kT = -0.00000001;
						break;

						}
					
 						//now forces are computed, move nodes.
						 double beta;
						
						 /*for (int k = 0; k < generalParams.maxNodeCount; k++){
							if (generalParams.nodes_in_upperhem[k] == 1 && generalParams.boundaries_in_upperhem[k] != 1){
								coordInfoVecs.nodeForceX[k] = 0.001*coordInfoVecs.nodeForceX[k];
								coordInfoVecs.nodeForceY[k] = 0.001*coordInfoVecs.nodeForceY[k];
								coordInfoVecs.nodeForceZ[k] = 0.001*coordInfoVecs.nodeForceZ[k];
							}
						}*/
						
						/*for (int k = 0; k < generalParams.maxNodeCount; k++){
						
							coordInfoVecs.nodeForceX[k] += generalParams.volume_spring_constant*coordInfoVecs.SurfaceNormalX[k];
							coordInfoVecs.nodeForceY[k] += generalParams.volume_spring_constant*coordInfoVecs.SurfaceNormalY[k];
							coordInfoVecs.nodeForceZ[k] += generalParams.volume_spring_constant*coordInfoVecs.SurfaceNormalZ[k];
						  }*/
						  
						  
						  
						//  std::cout<<"STOPPED BEFORE surfacenormal"<<std::endl;
						//std::cout<<"IS IT ADVANCE POSITION PROBLEM?"<<std::endl;
						/*for (int u = 0; u < generalParams.maxNodeCount; u++){
							int GAMMA = edgeswap_ptr->surfaceNormal_device_vecs(
								u,
								coordInfoVecs,
								generalParams
							);
						}*/
						//std::cout<<"STOPPED BEFORE AdvancePos"<<std::endl;
 						AdvancePositions(
 							coordInfoVecs,
 							generalParams,
							 domainParams);

						new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
					areaTriangleInfoVecs.area_triangle_energy + 
					bendingTriangleInfoVecs.bending_triangle_energy +
					0.5*energy_rep;// +
					//ljInfoVecs.lj_energy_M +  
					// ljInfoVecs.lj_energy_LJ +
						//generalParams.volume_energy;
				//std::cout<<"new_total_energy = "<<new_total_energy<<std::endl;

				energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy));
				old_total_energy = new_total_energy;
				current_time+=generalParams.dt;
				if (current_time >= Max_Runtime*0.25 && energy_gradient < energy_gradient_threshold){
					end_of_relaxation = true;
					
					if (end_of_relaxation == true){
						//	std::cout<<"SIMULATIONs TRIGGER REPOSITIONING AND EDGESWAP?"<<std::endl;

							newcenterX = 0.0;
							newcenterY = 0.0;
							newcenterZ = 0.0;
						//	std::cout<<"HERE?"<<std::endl;
							
							for (int i = 0; i < generalParams.maxNodeCount; i++){//for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
								//std::cout<<i<<std::endl;
								newcenterX += coordInfoVecs.nodeLocX[i];
								//std::cout<<newcenterX<<std::endl;
								newcenterY += coordInfoVecs.nodeLocY[i];
								//std::cout<<newcenterY<<std::endl;
								newcenterZ += coordInfoVecs.nodeLocZ[i];
								//std::cout<<newcenterZ<<std::endl;
							}
						//	std::cout<<"HERE2?"<<std::endl;
							newcenterX = newcenterX/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
							newcenterY = newcenterY/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
							newcenterZ = newcenterZ/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
							displacementX = newcenterX - generalParams.centerX;
							displacementY = newcenterY - generalParams.centerY;
							displacementZ = newcenterZ - generalParams.centerZ;
							
						//	std::cout<<"HERE3?"<<std::endl;
							for (int i = 0; i < generalParams.maxNodeCount; i++){
							coordInfoVecs.nodeLocX[i] += -displacementX;
							coordInfoVecs.nodeLocY[i] += -displacementY;
							coordInfoVecs.nodeLocZ[i] += -displacementZ;
							}
						//	std::cout<<"HERE4?"<<std::endl;
							for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
								ljInfoVecs.LJ_PosX_all[i] += -displacementX;
								ljInfoVecs.LJ_PosY_all[i] += -displacementY;
								ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
							}

						//	std::cout<<"HERE5?"<<std::endl;
						/*	int maxElementIndex = std::max_element(coordInfoVecs.nodeLocZ.begin(),coordInfoVecs.nodeLocZ.end()) - coordInfoVecs.nodeLocZ.begin();
							for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
								if (i == 0){
									filament_base[i] = maxElementIndex;
									continue;
								}
								for (int j = 0; j < generalParams.maxNodeCount; j++){
									double xsquared = (ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j])*
														(ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j]);
									double ysquared = (ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j])*
														(ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j]);
									double zsquared = (ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j])*
														(ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j]);
									double R = sqrt(xsquared + ysquared + zsquared);
									if (R < (max_height - min_height)/2.0 && j != maxElementIndex){
										filament_base[i] = j;
										break;
									}
									else{filament_base[i] = -1;}
								}
							} */
						//	std::cout<<"ERROR 0"<<std::endl;

							ComputeVolume(
								generalParams,
								coordInfoVecs,
								linearSpringInfoVecs,
								ljInfoVecs);
							//std::cout<<"ERROR 1"<<std::endl;
							
							//std::cout<<"BEGIN EDGESWAP ALGORITHM"<<std::endl;
							edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
							//std::cout<<"ERROR 1.5"<<std::endl;
							VectorShuffleForEdgeswapLoop.clear();
							for (int i = 0; i < coordInfoVecs.num_edges; i++){
								if (generalParams.edges_in_upperhem_list[i] >= 0 && 
									generalParams.edges_in_upperhem_list[i] != INT_MAX &&
									generalParams.boundaries_in_upperhem[i] != 1)
									VectorShuffleForEdgeswapLoop.push_back(generalParams.edges_in_upperhem_list[i]);
								}	
						//	std::cout<<"STOPPED BEFORE edgeswap"<<std::endl;
							//std::random_device rand_dev;
							//std::mt19937 generator_edgeswap(rand_dev());
							num_edge_loop = round(true_num_edges_in_upperhem*SAMPLE_SIZE);
							if (num_edge_loop == 0){
								num_edge_loop = 1;
							}
							//generalParams.kT = generalParams.kT*2.0;
						//	double kT_reduction = generalParams.kT/5.0;
						//while (generalParams.kT > 0.15){
							std::shuffle(std::begin(VectorShuffleForEdgeswapLoop), std::end(VectorShuffleForEdgeswapLoop), generator_edgeswap);
							//	std::shuffle(std::begin(generalParams.edges_in_upperhem_list), std::end(generalParams.edges_in_upperhem_list), generator_edgeswap);
								//for (int edge_loop = 0; edge_loop < VectorShuffleForEdgeswapLoop.size(); edge_loop++){
								for (int edge_loop = 0; edge_loop < num_edge_loop; edge_loop++) {
									//std::cout<<"edge_loop = "<<edge_loop<<std::endl;
									
									//std::random_device rand_dev;
									//std::mt19937 generator(rand_dev());
								
								std::uniform_int_distribution<int> distribution(1,VectorShuffleForEdgeswapLoop.size());
								
								int dice_roll = distribution(generator_edgeswap);
								
								int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
								//int edge = dice_roll -1;
								while (generalParams.boundaries_in_upperhem[edge] == 1 || edge == INT_MAX || edge < 0){
										dice_roll = distribution(generator_edgeswap);
										
										edge =  generalParams.edges_in_upperhem_list[dice_roll - 1];
										edge = dice_roll -1;
									 }
									//int edge = generalParams.edges_in_upperhem_list[edge_loop];
									//int edge = VectorShuffleForEdgeswapLoop[edge_loop];
									//std::cout<<"edge = "<<edge<<std::endl;
									if (edge < 0 || edge == INT_MAX){
										continue;
									}

									int ALPHA = edgeswap_ptr->edge_swap_host_vecs(
										edge,
										generalParams,
										build_ptr->hostSetInfoVecs,
										linearSpringInfoVecs,
										bendingTriangleInfoVecs,
										areaTriangleInfoVecs);
									
								}
							//	std::cout<<"STOPPED after edgeswap"<<std::endl;
						//		generalParams.kT -= kT_reduction;
						//	}
							//generalParams.kT = initial_kT;

						//std::cout<<"IS IT NODES2TRIANGLES PROBLEM?"<<std::endl;
						/*for (int u = 0; u < generalParams.maxNodeCount; u++){
							int BETA = edgeswap_ptr->nodes2Triangles_host_vecs(
								u,
								build_ptr->hostSetInfoVecs,
								coordInfoVecs,
								generalParams,
								auxVecs);
						}*/
						//std::cout<<"IT IS NOT"<<std::endl;
							//NOTE: EDGESWAP ALGORITHM CURRENTLY IS WRITTEN TO ALLOW AT MOST 8 NEIGHBORING NODES PER NODE.
							//std::cout<<"edgeswap done!"<<std::endl;
							edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
							//std::cout<<"END EDGESWAP ALGORITHM"<<std::endl;
							//std::cout<<"ERROR 2.5"<<std::endl;  
							
							//DETERMINE THE NORMAL VECTORS OF EACH NODE
						
						EDGESWAP_ALGORITHM_TRIGGERED = true;

						}
						break;
				}
			
			// std::cout<<"STOPPED at the end of one time step in relaxation"<<std::endl;

						//	 std::cout<<"IT'S NOT"<<std::endl;
						if (translate_counter % translate_frequency == 0){// || end_of_relaxation == true){
						//	std::cout<<"SIMULATIONs TRIGGER REPOSITIONING AND EDGESWAP?"<<std::endl;

							newcenterX = 0.0;
							newcenterY = 0.0;
							newcenterZ = 0.0;
						//	std::cout<<"HERE?"<<std::endl;
							
							for (int i = 0; i < generalParams.maxNodeCount; i++){//for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
								//std::cout<<i<<std::endl;
								newcenterX += coordInfoVecs.nodeLocX[i];
								//std::cout<<newcenterX<<std::endl;
								newcenterY += coordInfoVecs.nodeLocY[i];
								//std::cout<<newcenterY<<std::endl;
								newcenterZ += coordInfoVecs.nodeLocZ[i];
								//std::cout<<newcenterZ<<std::endl;
							}
						//	std::cout<<"HERE2?"<<std::endl;
							newcenterX = newcenterX/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
							newcenterY = newcenterY/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
							newcenterZ = newcenterZ/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
							displacementX = newcenterX - generalParams.centerX;
							displacementY = newcenterY - generalParams.centerY;
							displacementZ = newcenterZ - generalParams.centerZ;
							
						//	std::cout<<"HERE3?"<<std::endl;
							for (int i = 0; i < generalParams.maxNodeCount; i++){
							coordInfoVecs.nodeLocX[i] += -displacementX;
							coordInfoVecs.nodeLocY[i] += -displacementY;
							coordInfoVecs.nodeLocZ[i] += -displacementZ;
							}
						//	std::cout<<"HERE4?"<<std::endl;
							for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
								ljInfoVecs.LJ_PosX_all[i] += -displacementX;
								ljInfoVecs.LJ_PosY_all[i] += -displacementY;
								ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
							}

						//	std::cout<<"HERE5?"<<std::endl;
						/*	int maxElementIndex = std::max_element(coordInfoVecs.nodeLocZ.begin(),coordInfoVecs.nodeLocZ.end()) - coordInfoVecs.nodeLocZ.begin();
							for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
								if (i == 0){
									filament_base[i] = maxElementIndex;
									continue;
								}
								for (int j = 0; j < generalParams.maxNodeCount; j++){
									double xsquared = (ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j])*
														(ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j]);
									double ysquared = (ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j])*
														(ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j]);
									double zsquared = (ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j])*
														(ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j]);
									double R = sqrt(xsquared + ysquared + zsquared);
									if (R < (max_height - min_height)/2.0 && j != maxElementIndex){
										filament_base[i] = j;
										break;
									}
									else{filament_base[i] = -1;}
								}
							} */
						//	std::cout<<"ERROR 0"<<std::endl;

							ComputeVolume(
								generalParams,
								coordInfoVecs,
								linearSpringInfoVecs,
								ljInfoVecs);
							//std::cout<<"ERROR 1"<<std::endl;
							
							//std::cout<<"BEGIN EDGESWAP ALGORITHM"<<std::endl;
							edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
							//std::cout<<"ERROR 1.5"<<std::endl;
							VectorShuffleForEdgeswapLoop.clear();
							for (int i = 0; i < coordInfoVecs.num_edges; i++){
								if (generalParams.edges_in_upperhem_list[i] >= 0 && 
									generalParams.edges_in_upperhem_list[i] != INT_MAX &&
									generalParams.boundaries_in_upperhem[i] != 1)
									VectorShuffleForEdgeswapLoop.push_back(generalParams.edges_in_upperhem_list[i]);
								}	
						//	std::cout<<"STOPPED BEFORE edgeswap"<<std::endl;
							//std::random_device rand_dev;
							//std::mt19937 generator_edgeswap(rand_dev());
							num_edge_loop = round(true_num_edges_in_upperhem*SAMPLE_SIZE);
							if (num_edge_loop == 0){
								num_edge_loop = 1;
							}
							//generalParams.kT = generalParams.kT*2.0;
						//	double kT_reduction = generalParams.kT/5.0;
						//while (generalParams.kT > 0.15){
							std::shuffle(std::begin(VectorShuffleForEdgeswapLoop), std::end(VectorShuffleForEdgeswapLoop), generator_edgeswap);
							//	std::shuffle(std::begin(generalParams.edges_in_upperhem_list), std::end(generalParams.edges_in_upperhem_list), generator_edgeswap);
								//for (int edge_loop = 0; edge_loop < VectorShuffleForEdgeswapLoop.size(); edge_loop++){
								for (int edge_loop = 0; edge_loop < num_edge_loop; edge_loop++) {
									//std::cout<<"edge_loop = "<<edge_loop<<std::endl;
									
									//std::random_device rand_dev;
									//std::mt19937 generator(rand_dev());
								
								std::uniform_int_distribution<int> distribution(1,VectorShuffleForEdgeswapLoop.size());
								
								int dice_roll = distribution(generator_edgeswap);
								
								int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
								//int edge = dice_roll -1;
								while (generalParams.boundaries_in_upperhem[edge] == 1 || edge == INT_MAX || edge < 0){
										dice_roll = distribution(generator_edgeswap);
										
										edge =  generalParams.edges_in_upperhem_list[dice_roll - 1];
										edge = dice_roll -1;
									 }
									//int edge = generalParams.edges_in_upperhem_list[edge_loop];
									//int edge = VectorShuffleForEdgeswapLoop[edge_loop];
									//std::cout<<"edge = "<<edge<<std::endl;
									if (edge < 0 || edge == INT_MAX){
										continue;
									}

									int ALPHA = edgeswap_ptr->edge_swap_host_vecs(
										edge,
										generalParams,
										build_ptr->hostSetInfoVecs,
										linearSpringInfoVecs,
										bendingTriangleInfoVecs,
										areaTriangleInfoVecs);
									
								}
							//	std::cout<<"STOPPED after edgeswap"<<std::endl;
						//		generalParams.kT -= kT_reduction;
						//	}
							//generalParams.kT = initial_kT;

						//std::cout<<"IS IT NODES2TRIANGLES PROBLEM?"<<std::endl;
						/*for (int u = 0; u < generalParams.maxNodeCount; u++){
							int BETA = edgeswap_ptr->nodes2Triangles_host_vecs(
								u,
								build_ptr->hostSetInfoVecs,
								coordInfoVecs,
								generalParams,
								auxVecs);
						}*/
						//std::cout<<"IT IS NOT"<<std::endl;
							//NOTE: EDGESWAP ALGORITHM CURRENTLY IS WRITTEN TO ALLOW AT MOST 8 NEIGHBORING NODES PER NODE.
							//std::cout<<"edgeswap done!"<<std::endl;
							edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
							//std::cout<<"END EDGESWAP ALGORITHM"<<std::endl;
							//std::cout<<"ERROR 2.5"<<std::endl;  
							
							//DETERMINE THE NORMAL VECTORS OF EACH NODE
						
						EDGESWAP_ALGORITHM_TRIGGERED = true;

						}
						

					
					if (generalParams.SCALE_TYPE != 3){
						if (translate_counter % (translate_frequency*5) == 1){
							max_height = -10000.0;
							for (int k = 0; k < generalParams.maxNodeCount; k++){
								if (coordInfoVecs. nodeLocZ[k] >= max_height){
									max_height = coordInfoVecs.nodeLocZ[k];
									max_height_index = k;
								}
						
							}
							//std::cout<<"max_height_index = "<<max_height_index<<std::endl;
							dtb = 0.0;//dtb := distance to boundary
							generalParams.septin_ring_z = 0.0;
							generalParams.boundary_z = 0.0;
							//for (int k = 0; k < boundary_edge_list.size(); k++){
							for (int k = 0; k < boundary_node_list.size(); k++){
								double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
								//double n2 = coordInfoVecs.edges2Nodes_2[boundary_edge_list[k]];
								//double cent_of_edge_x = (coordInfoVecs.nodeLocX[n1] + coordInfoVecs.nodeLocX[n2])/2.0;
								//double cent_of_edge_y = (coordInfoVecs.nodeLocY[n1] + coordInfoVecs.nodeLocY[n2])/2.0;
								//double cent_of_edge_z = (coordInfoVecs.nodeLocZ[n1] + coordInfoVecs.nodeLocZ[n2])/2.0;
								double dist_x = coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1];//cent_of_edge_x;
								double dist_y = coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1];//cent_of_edge_y;
								double dist_z = coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1];//cent_of_edge_z;
								double temp_dist = sqrt((coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1])*(coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1]) +
								(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1])*(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1]) +
									(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1])*(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1]));
								generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
								if (temp_dist >= dtb){
									dtb = temp_dist;
									/* "dtb" will be used to identify where the septin ring is located, and used to determine the Hill coefficient*/
								}
							}
							//std::cout<<"dtb = "<<dtb<<std::endl;
							generalParams.septin_ring_z = generalParams.septin_ring_z/boundary_node_list.size();
							generalParams.boundary_z = generalParams.septin_ring_z - generalParams.Rmin;
							/* dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift */
							dtb_max = dtb + (generalParams.Rmin);
							// generalParams.septin_ring_z = 0.0;
							// generalParams.boundary_z = 0.0;
							// //for (int k = 0; k < boundary_edge_list.size(); k++){
							// for (int k = 0; k < boundary_node_list.size(); k++){
							// 	double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
							// 	generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
							// }
							//generalParams.septin_ring_z = generalParams.septin_ring_z/boundary_node_list.size();
							//generalParams.boundary_z = generalParams.septin_ring_z - generalParams.Rmin;
							/* dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift */
					
							generalParams.hilleqnconst = (dtb + generalParams.Rmin/4.0)/dtb_max;

							edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
							edgeswap_ptr->gradient_weakening_update_host_vecs(sigma,
								max_height_index,
								dtb,
								dtb_max,
								generalParams,
								coordInfoVecs,
								build_ptr->hostSetInfoVecs);
							edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
							}
					}	

				}
					//std::cout<<"current_time (# of relaxation step) = "<<current_time<<std::endl;
					if (EDGESWAP_ALGORITHM_TRIGGERED == false){
						//EDGE_SWAP IS TRIGGERED HERE IF THE RELAXATION IN THE PREVIOUS SECTION DID NOT HIT THE THRESHOLD VALUE TO TRIGGER
						//EDGESWAP NORMALLY.
						edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
							//std::cout<<"ERROR 1.5"<<std::endl;
							VectorShuffleForEdgeswapLoop.clear();
							for (int i = 0; i < coordInfoVecs.num_edges; i++){
								if (generalParams.edges_in_upperhem_list[i] >= 0 && 
									generalParams.edges_in_upperhem_list[i] != INT_MAX &&
									generalParams.boundaries_in_upperhem[i] != 1)
									VectorShuffleForEdgeswapLoop.push_back(generalParams.edges_in_upperhem_list[i]);
								}	
						//	std::cout<<"STOPPED BEFORE edgeswap"<<std::endl;
							//std::random_device rand_dev;
							//std::mt19937 generator_edgeswap(rand_dev());
							num_edge_loop = round(true_num_edges_in_upperhem*SAMPLE_SIZE);
							if (num_edge_loop == 0){
								num_edge_loop = 1;
							}
							//generalParams.kT = generalParams.kT*2.0;
						//	double kT_reduction = generalParams.kT/5.0;
						//while (generalParams.kT > 0.15){
							std::shuffle(std::begin(VectorShuffleForEdgeswapLoop), std::end(VectorShuffleForEdgeswapLoop), generator_edgeswap);
							//	std::shuffle(std::begin(generalParams.edges_in_upperhem_list), std::end(generalParams.edges_in_upperhem_list), generator_edgeswap);
								//for (int edge_loop = 0; edge_loop < VectorShuffleForEdgeswapLoop.size(); edge_loop++){
								for (int edge_loop = 0; edge_loop < num_edge_loop; edge_loop++) {
									//std::cout<<"edge_loop = "<<edge_loop<<std::endl;
									
									//std::random_device rand_dev;
									//std::mt19937 generator(rand_dev());
								
								std::uniform_int_distribution<int> distribution(1,VectorShuffleForEdgeswapLoop.size());
								
								int dice_roll = distribution(generator_edgeswap);
								
								int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
								//int edge = dice_roll -1;
								while (generalParams.boundaries_in_upperhem[edge] == 1 || edge == INT_MAX || edge < 0){
										dice_roll = distribution(generator_edgeswap);
										
										edge =  generalParams.edges_in_upperhem_list[dice_roll - 1];
										edge = dice_roll -1;
									 }
									//int edge = generalParams.edges_in_upperhem_list[edge_loop];
									//int edge = VectorShuffleForEdgeswapLoop[edge_loop];
									//std::cout<<"edge = "<<edge<<std::endl;
									if (edge < 0 || edge == INT_MAX){
										continue;
									}

									int ALPHA = edgeswap_ptr->edge_swap_host_vecs(
										edge,
										generalParams,
										build_ptr->hostSetInfoVecs,
										linearSpringInfoVecs,
										bendingTriangleInfoVecs,
										areaTriangleInfoVecs);
									
								}
							//	std::cout<<"STOPPED after edgeswap"<<std::endl;
						//		generalParams.kT -= kT_reduction;
						//	}
							//generalParams.kT = initial_kT;

						//std::cout<<"IS IT NODES2TRIANGLES PROBLEM?"<<std::endl;
						/*for (int u = 0; u < generalParams.maxNodeCount; u++){
							int BETA = edgeswap_ptr->nodes2Triangles_host_vecs(
								u,
								build_ptr->hostSetInfoVecs,
								coordInfoVecs,
								generalParams,
								auxVecs);
						}*/
						//std::cout<<"IT IS NOT"<<std::endl;
							//NOTE: EDGESWAP ALGORITHM CURRENTLY IS WRITTEN TO ALLOW AT MOST 8 NEIGHBORING NODES PER NODE.
							//std::cout<<"edgeswap done!"<<std::endl;
							edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
					}
					//std::cout<<"energy_gradient = "<<energy_gradient<<std::endl;
					//std::cout<<"end relaxation step"<<std::endl;
					 
						
			
 					/*if (edgeswap_iteration % (2*RECORD_TIME) == 0){
						if (reduce_counter*0.05 < 0.89){
							reduce_counter = reduce_counter + 1;
						linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant - linearSpringInfoVecs.spring_constant*(reduce_counter*0.05);
						bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant - bendingTriangleInfoVecs.spring_constant*(reduce_counter*0.05);
						areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant - areaTriangleInfoVecs.spring_constant*(reduce_counter*0.05);
						std::cout<<"current weakened linear spring coeff = "<<linearSpringInfoVecs.spring_constant_weak<<std::endl;
						std::cout<<"current weakened bending spring ceoff = "<<bendingTriangleInfoVecs.spring_constant_weak<<std::endl;
						std::cout<<"current weakened area spring coeff = "<<areaTriangleInfoVecs.spring_constant_weak<<std::endl;
						}
						else{}
					 }*/		
					
 					if (edgeswap_iteration % RECORD_TIME == 0){

						for (int v = 0; v < coordInfoVecs.num_edges; v++){
							double ev1 = coordInfoVecs.edges2Nodes_1[v];
							double ev2 = coordInfoVecs.edges2Nodes_2[v];
							if (ev1 == INT_MAX || ev2 == INT_MAX){
								continue;
							}
							double ed = sqrt((coordInfoVecs.nodeLocX[ev2] - coordInfoVecs.nodeLocX[ev1])*(coordInfoVecs.nodeLocX[ev2] - coordInfoVecs.nodeLocX[ev1]) +
										(coordInfoVecs.nodeLocY[ev2] - coordInfoVecs.nodeLocY[ev1])*(coordInfoVecs.nodeLocY[ev2] - coordInfoVecs.nodeLocY[ev1]) +
										(coordInfoVecs.nodeLocZ[ev2] - coordInfoVecs.nodeLocZ[ev1])*(coordInfoVecs.nodeLocZ[ev2] - coordInfoVecs.nodeLocZ[ev1]));
							if (ed >= 2.0){
								std::cout<<"Edge over extension, possibly instability occuring"<<std::endl;
								runSim = false;
								initial_kT = -0.00000000001;
								break;
							}
						}
						generalParams.angle_per_edge.clear();
						//generalParams.angle_per_edge.resize(coordInfoVecs.num_edges);
						int j = 0;
						for (int j = 0; j < coordInfoVecs.num_edges; j++){
							if (coordInfoVecs.edges2Nodes_1[j] == INT_MAX || coordInfoVecs.edges2Nodes_2[j] == INT_MAX){
								generalParams.angle_per_edge.push_back(-INT_MAX);
								continue;
							}
							
							double T1 = coordInfoVecs.edges2Triangles_1[j];
							double T2 = coordInfoVecs.edges2Triangles_2[j];
							double T1v1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[T1]] - coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[T1]];
							double T1v1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[T1]] - coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[T1]];
							double T1v1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[T1]] - coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[T1]];
							double T1v2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[T1]] - coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[T1]];
							double T1v2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[T1]] - coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[T1]];
							double T1v2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[T1]] - coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[T1]];
							double T2v1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[T2]] - coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[T2]];
							double T2v1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[T2]] - coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[T2]];
							double T2v1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[T2]] - coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[T2]];
							double T2v2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[T2]] - coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[T2]];
							double T2v2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[T2]] - coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[T2]];
							double T2v2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[T2]] - coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[T2]];
							double N1x = T1v1y*T1v2z - T1v2y*T1v1z;
							double N1y = -(T1v1x*T1v2z - T1v2x*T1v1z);
							double N1z = T1v1x*T1v2y - T1v2x*T1v1y;
							double N2x = T2v1y*T2v2z - T2v2y*T2v1z;
							double N2y = -(T2v1x*T2v2z - T2v2x*T2v1z);
							double N2z = T2v1x*T2v2y - T2v2x*T2v1y;
							
							double nN1 = sqrt(N1x*N1x + N1y*N1y + N1z*N1z);
							double nN2 = sqrt(N2x*N2x + N2y*N2y + N2z*N2z);
							double cosAngle = (N1x*N2x + N1y*N2y + N1z*N2z)/ (nN1*nN2);

							double direction_check_x = N1y*N2z - N2y*N1z;
							double direction_check_y = -(N1x*N2z - N2x*N1z);
							double direction_check_z = (N1x*N2y - N2x*N1y);
							double edge_direction_x = coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[j]] - coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[j]];
							double edge_direction_y = coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[j]] - coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[j]];
							double edge_direction_z = coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[j]] - coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[j]];
							double direction_check = direction_check_x*edge_direction_x + direction_check_y*edge_direction_y + direction_check_z*edge_direction_z;

							if (direction_check >= 0){
								generalParams.angle_per_edge.push_back( acos(cosAngle));
							}
							else{
								generalParams.angle_per_edge.push_back( -acos(cosAngle));
							}
							//j += 1;
							
						}
						generalParams.true_num_edges = 0;
						for (int i = 0; i < coordInfoVecs.num_edges; i++){
							if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
								generalParams.true_num_edges += 1;
							}
						 }
						 //std::cout<<"push_nodes_down size = "<<push_nodes_down.size()<<std::endl;
						 //std::cout<<"push_nodes_up size = "<<push_nodes_up.size()<<std::endl;
						// for (int i = 0; i < push_nodes_down.size(); i++){
						 //	std::cout<<"push_nodes_down "<<push_nodes_down[i]<<std::endl;
						 //}
						 //for (int i = 0; i < push_nodes_up.size(); i++){
				//			std::cout<<"push_nodes_up "<<push_nodes_up[i]<<std::endl;
						//}
						if (true){
							max_height = -10000.0;
							min_height = 10000.0;
							for (int k = 0; k < generalParams.maxNodeCount; k++){
								if (coordInfoVecs. nodeLocZ[k] >= max_height){
									max_height = coordInfoVecs. nodeLocZ[k];
								}
								if (coordInfoVecs.nodeLocZ[k] <= min_height){
									min_height = coordInfoVecs.nodeLocZ[k];
								}
							}
							std::cout<<"cell diameter = "<<max_height - min_height<<std::endl;
						}
						 storage->print_VTK_File();
						 //std::cout<<"current Hill equation constant = "<<generalParams.hilleqnconst<<std::endl;
						 //storage->storeVariables();
						 std::cout<<"current total energy = "<< new_total_energy<<std::endl;
						 std::cout<<"LINEAR ENERGY = "<<linearSpringInfoVecs.linear_spring_energy<<std::endl;
						std::cout<<"BEND ENERGY = "<<bendingTriangleInfoVecs.bending_triangle_energy<<std::endl;
						std::cout<<"AREA ENERGY = "<<areaTriangleInfoVecs.area_triangle_energy<<std::endl;
						std::cout<<"REPULSION ENERGY = "<<energy_rep<<std::endl;
						std::cout<<"VOLUME ENERGY = "<<generalParams.volume_energy<<std::endl;
						 std::cout<<"energy_gradient = "<<energy_gradient<<std::endl;
						 std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
						std::cout<<"equilibrium total volume = "<<generalParams.eq_total_volume<<std::endl;
 					}
 					if (edgeswap_iteration == NKBT-1 ){
 						//storage->storeVariables();
					 }

					

					 edgeswap_iteration += 1;
					 
 					if (edgeswap_iteration == NKBT){
 						generalParams.kT = -1.0;//generalParams.kT - 0.072;
 						std::cout<<"Current kBT = "<<generalParams.kT<<std::endl;
 						edgeswap_iteration = 0;
 					}
 					if (generalParams.kT < min_kT){
 						initial_kT = -1.0;
					runSim = false;
					break;
					 }

//std::cout<<"ERROR BEFORE GROWTH"<<std::endl;
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////// GROWTH OF THE CELL (MEMBRANE) ////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////	
if (edgeswap_iteration % GROWTH_FREQUENCY == 0){
VectorShuffleForGrowthLoop.clear();
int VectorShuffleForGrowthLoop_COUNT = 0;
for (int y = 0; y < coordInfoVecs.num_edges; y++){
	if (generalParams.edges_in_upperhem_list[y] >= 0 &&
		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
		generalParams.boundaries_in_upperhem[y] != 1){
		VectorShuffleForGrowthLoop.push_back(y);
		VectorShuffleForGrowthLoop_COUNT += 1;
	}
	/*if (generalParams.edges_in_upperhem_list[y] >= 0 &&
		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
		generalParams.boundaries_in_upperhem[y] != 1 &&
		edges_in_growth[y] == 1){
		VectorShuffleForGrowthLoop.push_back(y);
	}*/
	
	
}
std::cout<<VectorShuffleForGrowthLoop_COUNT<<std::endl;

//std::random_device rand_dev;
//std::mt19937 generator2(rand_dev());
std::shuffle(std::begin(VectorShuffleForGrowthLoop), std::end(VectorShuffleForGrowthLoop), generator2);
int MAX_GROWTH_TEST = VectorShuffleForGrowthLoop.size();
bool triggered = false;
int true_DELTA = 0;
//std::cout<<"BEGIN GROWTH ALGORITHM"<<std::endl;
edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
int GROWTH_COUNT = 0;
for (int p = 0; p < MAX_GROWTH_TEST; p++){
	//if (GROWTH_COUNT >= MAX_GROWTH_NUMBER){
		//std::cout<<"GROWTH_COUNT = "<<GROWTH_COUNT<<std::endl;
	//	break;	
	//}
	if (coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]] < 0 || coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]] == INT_MAX){
		continue;
	}
	else if (coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]] < 0 || coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]] == INT_MAX){
		continue;
	}
	//std::cout<<"begin growth test"<<std::endl;
	int DELTA = edgeswap_ptr->growth_host_vecs(
		VectorShuffleForGrowthLoop[p],
		generalParams,
		build_ptr->hostSetInfoVecs,
		coordInfoVecs,
		linearSpringInfoVecs,
		bendingTriangleInfoVecs,
		areaTriangleInfoVecs);
	GROWTH_COUNT += DELTA;
	//IN THIS CODE, THE GROWTH IS DETERMINISTIC SUCH THAT A CHOSEN EDGE WILL ALWAYS UNDERGO GROWTH!!!!!!!!!!!!!!!!!!!!!
	// std::cout<<"chosen edge mid point x = "<<(coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]]] +
	// coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]]])/2.0<<std::endl;
	// std::cout<<"chosen edge mid point y = "<<(coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]]] +
	// coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]]])/2.0<<std::endl;
	// std::cout<<"chosen edge mid point z = "<<(coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]]] +
	// coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]]])/2.0<<std::endl;
}
edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
//std::cout<<"END GROWTH ALGORITHM"<<std::endl;
if (true_DELTA >= 1){
	triggered = true;
	std::cout<<"number of cell wall insertion = "<<true_DELTA<<std::endl;
	storage->print_VTK_File();
	std::cout<<"cell wall insertion triggered!"<<std::endl;
}

				if (triggered == true){	
					true_num_edges_in_upperhem = 0;
					for (int i = 0; i < coordInfoVecs.num_edges; i++){
						if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
							true_num_edges_in_upperhem += 1;
							//break;
						}
					}
					//std::cout<<"WHERE iS THE PROBLEM 3"<<std::endl;
				}
			}
			
			
			
			
//std::cout<<"GROWTH DONE!"<<std::endl;
 ////storage->print_VTK_File();
////storage->storeVariables();

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////// END OF GROWTH SECTION //////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					

ComputeVolume(
	generalParams,
	coordInfoVecs,
	linearSpringInfoVecs,
	ljInfoVecs);
					
					
 			}
		
		}
		

	};
	
	





void System::assignStorage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
};
void System::set_weak_builder(std::weak_ptr<SystemBuilder> _weak_bld_ptr) {
	weak_bld_ptr = _weak_bld_ptr;
};



//initialize memory for thrust vectors and set coordInfoVecs vals from input. 
void System::initializeSystem(HostSetInfoVecs& hostSetInfoVecs) {
	std::cout<<"Initializing"<<std::endl;

	generalParams.maxNodeCount = hostSetInfoVecs.nodeLocX.size();
	coordInfoVecs.num_edges = hostSetInfoVecs.edges2Nodes_1.size();
	coordInfoVecs.num_triangles = hostSetInfoVecs.triangles2Nodes_1.size();

	std::cout<<"num nodes: "<< generalParams.maxNodeCount << std::endl;
	std::cout<<"num edges: "<< coordInfoVecs.num_edges << std::endl;
	std::cout<<"num elems: "<< coordInfoVecs.num_triangles << std::endl;
	//allocate memory
	int mem_prealloc = 3;
	coordInfoVecs.scaling_per_edge.resize(mem_prealloc*coordInfoVecs.num_edges, 0.0);
	hostSetInfoVecs.scaling_per_edge.resize(coordInfoVecs.scaling_per_edge.size(), 0.0);

	coordInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(),false);
	coordInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	coordInfoVecs.nodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.nodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);

	coordInfoVecs.triangles2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_3.resize( mem_prealloc*coordInfoVecs.num_triangles );
	
	coordInfoVecs.triangles2Edges_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_3.resize( mem_prealloc*coordInfoVecs.num_triangles );

	coordInfoVecs.edges2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_edges );
	
	coordInfoVecs.edges2Triangles_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Triangles_2.resize( mem_prealloc*coordInfoVecs.num_edges );

	coordInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	coordInfoVecs.SurfaceNormalX.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.SurfaceNormalY.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.SurfaceNormalZ.resize( mem_prealloc*generalParams.maxNodeCount);

	generalParams.nodes_in_upperhem.resize(mem_prealloc*generalParams.maxNodeCount);
	generalParams.triangles_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_triangles);
	generalParams.edges_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges);
	generalParams.edges_in_upperhem_list.resize(mem_prealloc*coordInfoVecs.num_edges);
	generalParams.boundaries_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges, -1);

	hostSetInfoVecs.nodes_in_upperhem.resize(generalParams.nodes_in_upperhem.size());
	hostSetInfoVecs.triangles_in_upperhem.resize(generalParams.triangles_in_upperhem.size());
	hostSetInfoVecs.edges_in_upperhem.resize(generalParams.edges_in_upperhem.size());
	hostSetInfoVecs.edges_in_upperhem_list.resize(mem_prealloc*coordInfoVecs.num_edges);
	hostSetInfoVecs.boundaries_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges, -1);

	hostSetInfoVecs.nodes2Triangles_1.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_2.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_3.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_4.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_5.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_6.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_7.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_8.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_9.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	

	coordInfoVecs.nodes2Triangles_1.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_2.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_3.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_4.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_5.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_6.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_7.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_8.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_9.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	

	thrust::copy(coordInfoVecs.nodes2Triangles_1.begin(), coordInfoVecs.nodes2Triangles_1.end(), hostSetInfoVecs.nodes2Triangles_1.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_2.begin(), coordInfoVecs.nodes2Triangles_2.end(), hostSetInfoVecs.nodes2Triangles_2.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_3.begin(), coordInfoVecs.nodes2Triangles_3.end(), hostSetInfoVecs.nodes2Triangles_3.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_4.begin(), coordInfoVecs.nodes2Triangles_4.end(), hostSetInfoVecs.nodes2Triangles_4.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_5.begin(), coordInfoVecs.nodes2Triangles_5.end(), hostSetInfoVecs.nodes2Triangles_5.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_6.begin(), coordInfoVecs.nodes2Triangles_6.end(), hostSetInfoVecs.nodes2Triangles_6.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_7.begin(), coordInfoVecs.nodes2Triangles_7.end(), hostSetInfoVecs.nodes2Triangles_7.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_8.begin(), coordInfoVecs.nodes2Triangles_8.end(), hostSetInfoVecs.nodes2Triangles_8.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_9.begin(), coordInfoVecs.nodes2Triangles_9.end(), hostSetInfoVecs.nodes2Triangles_9.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_10.begin(), coordInfoVecs.nodes2Triangles_10.end(), hostInfoVecs.nodes2Triangles_10.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_11.begin(), coordInfoVecs.nodes2Triangles_11.end(), hostInfoVecs.nodes2Triangles_11.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_12.begin(), coordInfoVecs.nodes2Triangles_12.end(), hostInfoVecs.nodes2Triangles_12.begin() );

	//copy info to GPU
	std::cout<<"Copying"<<std::endl;
	thrust::copy(hostSetInfoVecs.isNodeFixed.begin(),hostSetInfoVecs.isNodeFixed.end(), coordInfoVecs.isNodeFixed.begin());
	
	std::cout<<"fixed_node_in_host: "<<std::endl;
	for (int k = 0; k < hostSetInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<hostSetInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_host_printout"<<std::endl;
	std::cout<<"fixed_node_in_device: "<<std::endl;
	for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<coordInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_device_printout"<<std::endl;
std::cout<<"size of host fixed "<< hostSetInfoVecs.isNodeFixed.size()<<std::endl;
std::cout<<"size of device fixed "<< coordInfoVecs.isNodeFixed.size()<<std::endl;

	/*for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		bool isFixedHost = hostSetInfoVecs.isNodeFixed[k];
		bool isFixedDevice = coordInfoVecs.isNodeFixed[k];
		if (isFixedDevice != isFixedHost){

			std::cout<<"pos "<< k << " dev val = " << coordInfoVecs.isNodeFixed[k]
				<< " host val = " <<  hostSetInfoVecs.isNodeFixed[k] <<std::endl;
		}
	}*/
	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

	thrust::fill(coordInfoVecs.prevNodeForceX.begin(), coordInfoVecs.prevNodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceY.begin(), coordInfoVecs.prevNodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceZ.begin(), coordInfoVecs.prevNodeForceZ.end(), 0.0);
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.prevNodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.prevNodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.prevNodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.nodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.nodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.nodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Nodes_1.begin(), hostSetInfoVecs.triangles2Nodes_1.end(), coordInfoVecs.triangles2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_2.begin(), hostSetInfoVecs.triangles2Nodes_2.end(), coordInfoVecs.triangles2Nodes_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_3.begin(), hostSetInfoVecs.triangles2Nodes_3.end(), coordInfoVecs.triangles2Nodes_3.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Edges_1.begin(), hostSetInfoVecs.triangles2Edges_1.end(), coordInfoVecs.triangles2Edges_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_2.begin(), hostSetInfoVecs.triangles2Edges_2.end(), coordInfoVecs.triangles2Edges_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_3.begin(), hostSetInfoVecs.triangles2Edges_3.end(), coordInfoVecs.triangles2Edges_3.begin() );

	thrust::copy(hostSetInfoVecs.edges2Nodes_1.begin(), hostSetInfoVecs.edges2Nodes_1.end(), coordInfoVecs.edges2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Nodes_2.begin(), hostSetInfoVecs.edges2Nodes_2.end(), coordInfoVecs.edges2Nodes_2.begin() );
	
	thrust::copy(hostSetInfoVecs.edges2Triangles_1.begin(), hostSetInfoVecs.edges2Triangles_1.end(), coordInfoVecs.edges2Triangles_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Triangles_2.begin(), hostSetInfoVecs.edges2Triangles_2.end(), coordInfoVecs.edges2Triangles_2.begin() );

	thrust::copy(hostSetInfoVecs.nndata1.begin(), hostSetInfoVecs.nndata1.end(), coordInfoVecs.nndata1.begin() );
	thrust::copy(hostSetInfoVecs.nndata2.begin(), hostSetInfoVecs.nndata2.end(), coordInfoVecs.nndata2.begin() );
	thrust::copy(hostSetInfoVecs.nndata3.begin(), hostSetInfoVecs.nndata3.end(), coordInfoVecs.nndata3.begin() );
	thrust::copy(hostSetInfoVecs.nndata4.begin(), hostSetInfoVecs.nndata4.end(), coordInfoVecs.nndata4.begin() );
	thrust::copy(hostSetInfoVecs.nndata5.begin(), hostSetInfoVecs.nndata5.end(), coordInfoVecs.nndata5.begin() );
	thrust::copy(hostSetInfoVecs.nndata6.begin(), hostSetInfoVecs.nndata6.end(), coordInfoVecs.nndata6.begin() );
	thrust::copy(hostSetInfoVecs.nndata7.begin(), hostSetInfoVecs.nndata7.end(), coordInfoVecs.nndata7.begin() );
	thrust::copy(hostSetInfoVecs.nndata8.begin(), hostSetInfoVecs.nndata8.end(), coordInfoVecs.nndata8.begin() );
	thrust::copy(hostSetInfoVecs.nndata9.begin(), hostSetInfoVecs.nndata9.end(), coordInfoVecs.nndata9.begin() );
	//thrust::copy(hostSetInfoVecs.nndata10.begin(), hostSetInfoVecs.nndata10.end(), coordInfoVecs.nndata10.begin() );
	//thrust::copy(hostSetInfoVecs.nndata11.begin(), hostSetInfoVecs.nndata11.end(), coordInfoVecs.nndata11.begin() );
	//thrust::copy(hostSetInfoVecs.nndata12.begin(), hostSetInfoVecs.nndata12.end(), coordInfoVecs.nndata12.begin() );


 
	//allocate memory for other data structures.   

	//area triangle info vec
	//number of area springs is the number of triangles
	std::cout<<"Mem"<<std::endl;
	areaTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	
	areaTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);

	//beinding triangle info vec
	//num bending springs is the number of times each edge is between two triangles. 
	bendingTriangleInfoVecs.numBendingSprings = coordInfoVecs.num_edges;//coordInfoVecs.edges2Triangles_1.size();

	bendingTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	
	bendingTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);

	//linear springs
	
	linearSpringInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.tempNodeIdReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.edge_initial_length.clear();
	//linearSpringInfoVecs.edge_initial_length.resize(mem_prealloc*coordInfoVecs.num_edges,1.0);
	
	//thrust::copy(hostSetInfoVecs.edge_initial_length.begin(), hostSetInfoVecs.edge_initial_length.end(), linearSpringInfoVecs.edge_initial_length.begin() );

	//Resize the hostSetInfoVecs so that we can copy data back and forth between hostSetinfoVecs and coordInfoVecs without problem.
	hostSetInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	//hostSetInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	hostSetInfoVecs.nodeLocX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeLocX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.nodeForceX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeForceX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.triangles2Nodes_1.resize( coordInfoVecs.triangles2Nodes_1.size() );
	hostSetInfoVecs.triangles2Nodes_2.resize( coordInfoVecs.triangles2Nodes_2.size() );
	hostSetInfoVecs.triangles2Nodes_3.resize( coordInfoVecs.triangles2Nodes_3.size() );
	std::cout<<"Host_triangles2Nodes size = "<<hostSetInfoVecs.triangles2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.triangles2Edges_1.resize( coordInfoVecs.triangles2Edges_1.size() );
	hostSetInfoVecs.triangles2Edges_2.resize( coordInfoVecs.triangles2Edges_2.size() );
	hostSetInfoVecs.triangles2Edges_3.resize( coordInfoVecs.triangles2Edges_3.size() );
	std::cout<<"Host_triangles2Edges size = "<<hostSetInfoVecs.triangles2Edges_1.size()<<std::endl;

	hostSetInfoVecs.edges2Nodes_1.resize( coordInfoVecs.edges2Nodes_1.size() );
	hostSetInfoVecs.edges2Nodes_2.resize( coordInfoVecs.edges2Nodes_2.size() );
	std::cout<<"Host_edges2Nodes size = "<<hostSetInfoVecs.edges2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.edges2Triangles_1.resize( coordInfoVecs.edges2Triangles_1.size() );
	hostSetInfoVecs.edges2Triangles_2.resize( coordInfoVecs.edges2Triangles_2.size() );
	std::cout<<"Host_edges2Triangles size = "<<hostSetInfoVecs.edges2Triangles_1.size()<<std::endl;

	hostSetInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	//std::cout<<"initial lengths: "<< linearSpringInfoVecs.edge_initial_length.size()<<std::endl;

	std::cout<<"System Ready"<<std::endl;

	//Generate LJ particle list. and set LJ particle midpoint.
	//double maxX_lj = *(thrust::max_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double minX_lj = *(thrust::min_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double maxY_lj = *(thrust::max_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	//double minY_lj = *(thrust::min_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	
	//ljInfoVecs.LJ_PosX = (maxX_lj + minX_lj)/2.0;
	//ljInfoVecs.LJ_PosY = (maxY_lj + minY_lj)/2.0;


	//currently unused
	/*thrust::host_vector<int> tempIds;
	for (int i = 0; i < hostSetInfoVecs.nodeLocX.size(); i++ ) {
		double xLoc = hostSetInfoVecs.nodeLocX[i];
		double yLoc = hostSetInfoVecs.nodeLocY[i];
		double zLoc = hostSetInfoVecs.nodeLocZ[i];
		
		double xDist = ljInfoVecs.LJ_PosX - xLoc;
		double yDist = ljInfoVecs.LJ_PosY - yLoc;
		double zDist = ljInfoVecs.LJ_PosZ - zLoc;

		double dist = std::sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
		//just test all poitns for now. Optimize later.
		if (dist < ljInfoVecs.Rcutoff) {
			tempIds.push_back(i);
		}
	}
	ljInfoVecs.node_id_close.resize( tempIds.size() );
	thrust::copy(tempIds.begin(), tempIds.end(), ljInfoVecs.node_id_close.begin());
	std::cout<<"lj nodes: "<< ljInfoVecs.node_id_close.size() << std::endl;*/






	//last, set memory foor buckets.
	auxVecs.id_bucket.resize(generalParams.maxNodeCount);
	auxVecs.id_value.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded.resize(27 *( generalParams.maxNodeCount ));
 


};


